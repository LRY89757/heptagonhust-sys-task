#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <vector>
#include <cassert>
#include <numeric> 
#include <hip/hip_runtime.h>

#define PRINT_TIME(code) do { \
    auto start = system_clock::now(); \
    code \
    auto end   = system_clock::now(); \
    auto duration = duration_cast<microseconds>(end - start); \
    cout << "time spent: " << double(duration.count()) << "us" << endl; \
} while(0)

using namespace std;

using namespace chrono;

using vec = vector<int>; 

const int scale[] = {256, 512, 1024, 2048};
const string data_path("./data/");

// __global__ void Gemm(const int &size, vec &a, vec &b, vec &c)
__global__ void Gemm(const int &size, int *a, int *b, int *c)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int M = size, N = size, K=size;
    if(ty < M && tx < N) {
        int C = 0;
        for(int i = 0; i < K; ++i){
            C += a[ty * K + i] * b[i * N + tx];  // 1 FMA, 2 load, 计算访存比太低
        }
        c[ty * N + tx] = C;
    }
}

void CheckResult(const vec &c, const string &result_path)
{
    ifstream file_result(result_path);
    int nelems = c.size();
    float res_i;
    for (int i = 0; i < nelems; i++)
    {
        file_result >> res_i;
        assert(c[i] == res_i);
    }
    file_result.close();
}

// c = a * b
void Benchmark(const int &size)
{
    int dev = 0;
    hipSetDevice(dev);
    
    const int nelems = size * size;
    const string a_path(data_path + to_string(size) + "/a");
    const string b_path(data_path + to_string(size) + "/b");
    const string result_path(data_path + to_string(size) + "/result");
    ifstream file_a(a_path);
    ifstream file_b(b_path);

    vec a(nelems, 0);
    vec b(nelems, 0);
    vec c(nelems, 0);
    int *a_h, *b_h, *c_h;
    a_h = (int*)malloc(nelems * sizeof(int));
    b_h = (int*)malloc(nelems * sizeof(int));
    c_h = (int*)malloc(nelems * sizeof(int));

    for (int i = 0; i < nelems; i++)
    {
        file_a >> a[i];
        a_h[i] = a[i];
    }
    for (int i = 0; i < nelems; i++)
    {
        file_b >> b[i];
        b_h[i] = b[i];
    }


    int *a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, size*size*sizeof(int));
    hipMalloc((void**)&b_d, size*size*sizeof(int));
    hipMalloc((void**)&c_d, size*size*sizeof(int));

    //将host拷贝到device
    hipMemcpy((void*)a_d, (void*)a_h, nelems*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)b_d, (void*)b_h, nelems*sizeof(int), hipMemcpyHostToDevice);


    dim3 BlockSize(32, 32);
    dim3 GridSize((size+32-1)/32, (size+32-1)/32);
    // PRINT_TIME(
        // Gemm<<<GridSize, BlockSize>>>(size, a_d, a_d, c_d););
    Gemm<<<GridSize, BlockSize>>>(size, a_d, a_d, c_d);

    hipMemcpy((void*)c_h, (void*)c_d, nelems*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0;i<nelems;i++)
        c[i] = c_h[i];

    CheckResult(c, result_path);

    free(a_h);
    free(b_h);
    free(c_h);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    file_a.close();
    file_b.close();
}

int main()
{
    for (auto size : scale)
    {
        cout << "Running, dataset: size " << size << endl;
        Benchmark(size);
        cout << "Passed, dataset: size " << size << endl;
        cout << endl;
    }
    return 0;
}