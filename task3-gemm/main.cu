#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <vector>
#include <cassert>
#include <numeric> 
#include <hip/hip_runtime.h>

// nvcc main.cu -o cumain && sudo nvprof ./cumain

#define PRINT_TIME(code) do { \
    auto start = system_clock::now(); \
    code \
    auto end   = system_clock::now(); \
    auto duration = duration_cast<microseconds>(end - start); \
    cout << "time spent: " << double(duration.count()) << "us" << endl; \
} while(0)

using namespace std;

using namespace chrono;

using vec = vector<int>; 

const int scale[] = {256, 512, 1024, 2048};
const string data_path("./data/");

// __global__ void Gemm(const int &size, vec &a, vec &b, vec &c)
__global__ void Gemm(const int &size, const int *a, const int *b, int *c)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int M = size, N = size, K=size;
    if(ty < M && tx < N) {
        int ans = 0;
        for(int i = 0; i < K; ++i){
            ans += a[ty * K + i] * b[i * N + tx];  // 1 FMA, 2 load, 计算访存比太低
        }
        c[ty * N + tx] = ans;
        
        // c[1] = C;
        // C = 1;
    }
}

void CheckResult(const vec &c, const string &result_path)
{
    ifstream file_result(result_path);
    int nelems = c.size();
    float res_i;
    for (int i = 0; i < nelems; i++)
    {
        file_result >> res_i;
        // assert(c[i] == res_i);
    }
    file_result.close();
}

// c = a * b
void Benchmark(const int &size)
{


    const int nelems = size * size;
    const string a_path(data_path + to_string(size) + "/a");
    const string b_path(data_path + to_string(size) + "/b");
    const string result_path(data_path + to_string(size) + "/result");
    ifstream file_a(a_path);
    ifstream file_b(b_path);

    // vec a(nelems, 0);
    // vec b(nelems, 0);
    vec c(nelems, 0);
    int *a_h, *b_h, *c_h;
    a_h = (int*)malloc(nelems * sizeof(int));
    b_h = (int*)malloc(nelems * sizeof(int));
    c_h = (int*)malloc(nelems * sizeof(int));

    int *a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, nelems*sizeof(int));
    hipMalloc((void**)&b_d, nelems*sizeof(int));
    hipMalloc((void**)&c_d, nelems*sizeof(int));

    for (int i = 0; i < nelems; i++)
    {
        file_a >> a_h[i];
        // a_h[i] = a[i];
    }
    for (int i = 0; i < nelems; i++)
    {
        file_b >> b_h[i];
        // b_h[i] = b[i];
        // c_h[i] = 0;
    }

    //将host拷贝到device
    hipMemcpy((void*)a_d, (void*)a_h, nelems*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)b_d, (void*)b_h, nelems*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void*)c_d, (void*)c_h, nelems*sizeof(int), hipMemcpyHostToDevice);

    // cout<<nelems<<" "<<sizeof(c_d)<<"\n";
    // cout<<sizeof(a_h)<<"\n";


    dim3 BlockSize(32, 32);
    dim3 GridSize((size+BlockSize.x-1)/BlockSize.x, (size+BlockSize.y-1)/BlockSize.y);
    // PRINT_TIME(
        // Gemm<<<GridSize, BlockSize>>>(size, a_d, a_d, c_d););
    Gemm<<<GridSize, BlockSize>>>(size, a_d, b_d, c_d);

    hipMemcpy((void*)c_h, (void*)c_d, nelems*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0;i<nelems;i++)
        c[i] = c_h[i];

    CheckResult(c, result_path);

    free(a_h);
    free(b_h);
    free(c_h);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // cudaDeviceReset();
    hipError_t err = hipGetLastError(); // add
    if (err != hipSuccess)
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();

    file_a.close();
    file_b.close();
}

int main()
{
    int dev = 1;
    hipSetDevice(dev);

    for (auto size : scale)
    {
        cout << "Running, dataset: size " << size << endl;
        Benchmark(size);
        cout << "Passed, dataset: size " << size << endl;
        cout << endl;
    }
    return 0;
}