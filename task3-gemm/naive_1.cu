#include <iostream>
#include <hip/hip_runtime.h>

// nvcc naive_1.cu -o naive  && sudo nvprof ./naive
__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (ty < M && tx < N)
    {
        float c = 0;
        for (int i = 0; i < K; ++i)
        {
            c += A[ty * K + i] * B[i * N + tx]; // 1 FMA, 2 load, 计算访存比太低
        }
        C[ty * N + tx] = c;
    }
}

__global__ void matrixMul0(const float *A, const float *B, float *C,
                           int M, int N, int K)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (ty < M && tx < N)
    {
        for (int i = 0; i < K; ++i)
        {
            C[ty * N + tx] += A[ty * K + i] * B[i * N + tx];
        }
    }
}

__global__ void matrixMul1(const float *A, const float *B, float *C,
                           int M, int N, int K)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (ty < M && tx < N)
    {
        float c = 0;
        for (int i = 0; i < K; ++i)
        {
            c += A[tx * K + i] * B[i * N + ty];
        }
        C[tx * N + ty] = c;
    }
}

// __global__ void matrixMul2(const float *A, const float *B, float *C,
//                            int M, int N, int K)
// {
//     const int tx = threadIdx.x;
//     const int ty = threadIdx.y;
//     const int bx = blockIdx.x;
//     const int by = blockIdx.y;

//     int BLOCK = blockDim.x;
//     float *begin_a = (float*)A + by * BLOCK;
//     float *begin_b = (float*)B + bx * BLOCK;
//     float *end_a = begin_a + K;

//     float sum = 0.f;
//     for(float *a_ptr = begin_a, *b_ptr=begin_b;a_ptr<end_a;
//         a_ptr += BLOCK, b_ptr += BLOCK * n){

//         }

// }

// template <int BLOCK>
// __global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
//   float *c, int ldc) {
__global__ void sgemm(const float *a, const float *b, float *c, int m, int n, int k)
{
    int BLOCK = blockDim.x;
    int _m = blockIdx.x * BLOCK + threadIdx.x;
    int _n = blockIdx.y * BLOCK + threadIdx.y;
    if (_m < m && _n < n)
    {
        float sum = 0.f;
        for (int i = 0; i < k; ++i)
        {
            // sum += a[_m * k + i] * b[i * n + _n];
            sum += a[_n * k + i] * b[i * n + _m]; // 神奇了，就这行命令和上面那行命令一对比，居然可以快这么多
        }
        // c[_m * n + _n] = sum;
        c[_n * n + _m] = sum;
    }
}

template <int BLOCK>
// __global__ void sgemm1(int m, int n, int k, float *a, int lda, float *b, int ldb,
//   float *c, int ldc) {
__global__ void sgemm1(float *a, float *b, float *c, int m, int n, int k)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    float *begin_a = a + by * BLOCK * k;
    float *begin_b = b + bx * BLOCK;
    float *end_a = begin_a + k;

    float sum = 0.f;
    for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
         a_ptr += BLOCK, b_ptr += BLOCK * n)
    {
        __shared__ float ashare[BLOCK][BLOCK];
        __shared__ float bshare[BLOCK][BLOCK];

        ashare[ty][tx] = a_ptr[ty * k + tx];
        bshare[ty][tx] = b_ptr[ty * n + tx];
        __syncthreads();

#pragma unroll
        for (int kk = 0; kk < BLOCK; ++kk)
        {
            sum += ashare[ty][kk] * bshare[kk][tx];
        }
        __syncthreads();
    }

    c[(BLOCK * by + ty) * n + BLOCK * bx + tx] = sum;
}

// Host kernel Device Grid Block Thread
int main(int argc, char **argv)
{
    int dev = 0;
    hipSetDevice(dev);

    // 单个block最多可以调用多少个thread
    hipDeviceProp_t prop;
    // int devCnt = 0;
    hipGetDeviceProperties(&prop, 0);
    fprintf(stdout, "%d", prop.maxThreadsPerBlock);

    // int x1 = 8, x2=1<<18, x3=1024;
    // m, k, n
    int x1 = 1 << 11, x2 = x1, x3 = x1;

    //申请host内存
    float *a_h, *b_h, *dst_h;
    a_h = (float *)malloc(x1 * x2 * sizeof(float));
    b_h = (float *)malloc(x3 * x2 * sizeof(float));
    dst_h = (float *)malloc(x1 * x3 * sizeof(float));

    for (int i = 0; i < x1 * x2; ++i)
        a_h[i] = 1.0;

    for (int i = 0; i < x3 * x2; ++i)
        b_h[i] = 2.0;

    // 申请device内存
    float *a_d, *b_d, *dst_d;
    hipMalloc((void **)&a_d, x1 * x2 * sizeof(float));
    hipMalloc((void **)&b_d, x3 * x2 * sizeof(float));
    hipMalloc((void **)&dst_d, x1 * x3 * sizeof(float));

    //将host拷贝到device
    hipMemcpy((void *)a_d, (void *)a_h, x1 * x2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void *)b_d, (void *)b_h, x3 * x2 * sizeof(float), hipMemcpyHostToDevice);

    // dim3 grid(x1);
    // dim3 block(x3);
    dim3 blockSize(32, 32);
    dim3 gridSize((x1 + blockSize.x - 1) / blockSize.x,
                  (x3 + blockSize.y - 1) / blockSize.y);

    // Matrix_mul<<<grid, block>>>(a_d, b_d, dst_d, x1, x3, x2);
    // sgemm<<<gridSize, blockSize>>>(a_d, b_d, dst_d, x1, x3, x2);
    matrixMul0<<<gridSize, blockSize>>>(a_d, b_d, dst_d, x1, x3, x2);
    matrixMul<<<gridSize, blockSize>>>(a_d, b_d, dst_d, x1, x3, x2);
    matrixMul1<<<gridSize, blockSize>>>(a_d, b_d, dst_d, x1, x3, x2);
    // sgemm1<32><<<gridSize, blockSize>>>(a_d, b_d, dst_d, x1, x3, x2);

    // cudaDeviceSynchronize();
    // cudaMemcpy((void*)a_h, (void*)a_d, x1)
    hipMemcpy((void *)dst_h, (void *)dst_d, x1 * x3 * sizeof(float), hipMemcpyDeviceToHost);

    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < x1 * x3; ++i)
        maxError = fmax(maxError, fabs(dst_h[i] - 2 * x1));
    std::cout << "\n最大误差: " << maxError << std::endl;

    free(a_h);
    free(b_h);
    free(dst_h);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(dst_d);

    hipDeviceReset();
    return 0;
}
